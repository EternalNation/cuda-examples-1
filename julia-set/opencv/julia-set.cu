#include "hip/hip_runtime.h"
#include <boost/multi_array.hpp>
#include <iostream>
#include <omp.h>
#include <opencv2/opencv.hpp>
#include <complex>
#include "hip/hip_complex.h"

using namespace std;
using namespace cv;

// 2D Matrix typedef for pixel_map and the index
// Comment: likely should move this into a namespace
//  like bcuda::vector and bcuda::matrix or something
typedef boost::multi_array<int, 2> pixel_map;
typedef pixel_map::index pixel_map_index;

// Some operator overloads
__device__ __host__ hipDoubleComplex operator+(const hipDoubleComplex &a,
                                              const hipDoubleComplex &b) {
    return hipCadd(a, b);
}
__device__ __host__ hipDoubleComplex operator*(const hipDoubleComplex &a,
                                              const hipDoubleComplex &b) {
    return hipCmul(a, b);
}
__device__ __host__ double norm(const hipDoubleComplex &a) {
    return a.x * a.x + a.y * a.y;
}

__device__ int julia(const int &dimension, const int &x, const int &y) {
    const double scale = 1.5;
    const double jx = scale * static_cast<double>(dimension / 2 - x) /
        static_cast<double>(dimension / 2);
    const double jy = scale * static_cast<double>(dimension / 2 - y) /
        static_cast<double>(dimension / 2);
    hipDoubleComplex c = make_hipDoubleComplex(-0.8, 0.156);
    hipDoubleComplex a = make_hipDoubleComplex(jx, jy);
    for (size_t i = 0; i < 200; ++i) {
        //a = hipCadd(hipCmul(a, a), c);
        a = a * a + c;
        //if ((a.x * a.x + a.y + a.y) > 1000)
        if (norm(a) > 1000)
            return 0;
    }
    return 1;
}

// kernel to operate on pixel_map
__global__ void julia_set(const int *dimension, int *map) {
    const int x = blockIdx.x;
    const int y = blockIdx.y;
    const int offset = x + y * gridDim.x;
    map[offset] = julia(*dimension, x, y);
}

// The main function, we may parse command line
//  at some point for pixel_map size
int main(int argc, char **argv) {
    // Create the Matrix, likely substitute size from command line
    int size = 3000;
    pixel_map map(boost::extents[size][size]);

    // Zero out the pixel_map
#pragma omp parallel for    
    for (pixel_map_index i = 0; i < size; ++i) {
        for (pixel_map_index j = 0; j < size; ++j) {
            map[i][j] = 0;
        }
    }

    // Get the map to dev_map, and size to dev_size on the GPU
    int *dev_map;
    int *dev_size;
    hipMalloc((void **) &dev_map, map.num_elements() * sizeof(int));
    hipMalloc((void **) &dev_size, sizeof(int));
    hipMemcpy(dev_map, &map[0][0], map.num_elements() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
    dim3 grid(size, size);
    julia_set<<<grid, 1>>>(dev_size, dev_map);
    hipMemcpy(&map[0][0], dev_map, map.num_elements() * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_map);
    hipFree(dev_size);

    /*
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; ++j) {
            cout << i << ' ' << j << ' ' << test_julia(size, i, j) << '\n';
            map[i][j] = 255 * test_julia(size, i, j);
        }
    }
    */

    // Convert to OpenCV Image
    Mat image(size, size, CV_8UC3, Scalar(0, 0, 0));   
#pragma omp parallel for
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; ++j) {
            // Get color
            Vec3b color = image.at<Vec3b>(Point(i, j));
            // Mod color
            for (size_t k = 0; k < 3; ++k) {
                color[k] = 255 * map[i][j];
            }
            // Set color
            image.at<Vec3b>(Point(i, j)) = color;
        }
    }
    imwrite("julia-set.png", image);

/* CPU Code    
    // Populate pixel_map
#pragma omp parallel for
    for (pixel_map_index i = 0; i < size; ++i) {
        for (pixel_map_index j = 0; j < size; ++j) {
            map[i][j] = static_cast<complex<double> >(i + j) / static_cast<complex<double> >(2 * size);
        }
    }

    // Convert to OpenCV Image
    Mat image(size, size, CV_8UC3, Scalar(0, 0, 0));   
#pragma omp parallel for
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; ++j) {
            // Get color
            Vec3b color = image.at<Vec3b>(Point(i, j));
            // Mod color
            for (size_t k = 0; k < 3; ++k) {
                color[k] = static_cast<int>(map[i][j].real() * 255);
            }
            // Set color
            image.at<Vec3b>(Point(i, j)) = color;
        }
    }
    imwrite("julia-set.png", image);
 */
    return 0;
}
